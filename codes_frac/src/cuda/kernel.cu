#include "hip/hip_runtime.h"
#include "complex.h"
#include"kernel.cuh"
#include"kernel.h"
#include"whu_constants_and_definitions.h"
#include <assert.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include <hip/hip_runtime.h>
using namespace std;

#define CUDA_CALL(x) {const hipError_t a=(x); if(a != hipSuccess) {printf("\nerror in line:%d CUDAError:%s(err_num=%d)\n",__LINE__,hipGetErrorString(a),a); hipDeviceReset(); assert(0); }}

//const double rk=0.493677;

/*//const double rp=0.139556995;
#ifndef CAFFE_COMMON_CUH_
#define CAFFE_COMMON_CUH_


#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

#else
static __inline__ __device__ double atomicAdd(double *address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  if (val==0.0)
    return __longlong_as_double(old);
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}


#endif
#endif
*/
__device__ complex cro(
                       double sx,
                       double am1,
                       double am2) 
{
  double t1=(am1+am2) * (am1 + am2); // double t1=pow((am1+am2),2);
  double t2=(am1-am2) * (am1 - am2); // double t2=pow((am1-am2),2);
  double st=(sx-t1)*(sx-t2);
  double cro=sqrt(fabs(st))/sx;
  complex result;
  if (st<0.) result.y=cro;
  else result.x=cro;
  return  result;
}

__device__ complex propogator980(
                                 double mass,
                                 double g11,
                                 double g22,
                                 double sx)
{
  complex ci(0,1);
  double rm=mass*mass;
  complex propogator980=1.0/(rm-sx-ci*(g11*cro(sx,rp,rp)+g22*cro(sx,rk,rk)));
  return propogator980;
}

__device__ complex pip(
                       double sx)
{
  //?    complex ci(0,1);
  double xk2=sx-0.3116676;     //0.3116676=16.*0.139568*0.139568
  if(xk2<=0.)xk2=0.0;
  double r4pip=sqrt(xk2/sx)/(1.0+exp(9.8-3.5*sx));    //9.8=3.5*2.8
  return  make_complex(r4pip,0);
}

__device__ complex propogator(
                              double mass,
                              double width,
                              double sx) 
{
  complex ci(0,1);
  double am=mass;
  double g1=mass*width;
  complex prop=g1/(sx-am * am +ci*g1); // complex prop=g1/(sx-pow(am,2)+ci*g1);
  return prop;
}

__device__ complex propogator1270(
                                  double mass,
                                  double width,
                                  double sx) 
{
  complex ci(0,1);
  double rm=mass*mass;
  double gr=mass*width;
  double q2r=0.25*rm-0.0194792;
  double b2r=q2r*(q2r+0.1825)+0.033306;
  double g11270=gr*b2r/pow(q2r,2.5);
  double q2=0.25*sx-0.0194792;
  double b2=q2*(q2+0.1825)+0.033306;
  double g1=g11270*pow(q2,2.5)/b2;
  complex prop=gr/(sx-rm+ci*g1);
  return prop;
}
__global__ void cal_fCP(double *par, double *fCP_real, double *fCP_imag, int numbers)
{
  int id = threadIdx.x+blockDim.x*blockIdx.x;
  if(id<numbers){
    double rho0 = par[end_category * id + rho_category];
    double frac0 = par[end_category * id + frac_category];
    double phi0 = par[end_category * id + phi_category];
    rho0 *= exp(frac0);
    fCP_real[id]=rho0*cos(phi0);
    fCP_imag[id]=rho0*sin(phi0);
  }
}
__global__ void propogator1(
                            double mass,
                            double width,
                            double *sx,
                            double *b2qjvf2,
                            double *wu,
                            double *w0p22,
                            double *fCF_real,
                            double *fCF_imag,
                            int numbers)
{
  int id = threadIdx.x+blockDim.x*blockIdx.x;
  if(id<numbers){
    complex crp1 = propogator(mass, width, sx[id]);
    complex cr0p11 = crp1 / b2qjvf2[id];
    complex result;
    //01 contribution
    result = wu[id] * crp1;
    fCF_real[id] =result.x; 
    fCF_imag[id] =result.y; 
    result = wu[id+numbers] * crp1;
    fCF_real[id+numbers] =result.x; 
    fCF_imag[id+numbers] =result.y; 

    //02 contribution
    result = w0p22[id] * cr0p11;
    fCF_real[id+numbers*2] =result.x; 
    fCF_imag[id+numbers*2] =result.y; 
    result = w0p22[id+numbers] * cr0p11;
    fCF_real[id+numbers*3] =result.x; 
    fCF_imag[id+numbers*3] =result.y; 
  }

}
__global__ void propogator2(
                            double mass,
                            double g11,
                            double g22,
                            double *sx,
                            double *b2qjvf2,
                            double *wu,
                            double *w0p22,
                            double *fCF_real,
                            double *fCF_imag,
                            int numbers)
{
  int id = threadIdx.x+blockDim.x*blockIdx.x;
  if(id<numbers){
    id=threadIdx.x+blockDim.x*blockIdx.x;
    complex crp1 = propogator980(mass, g11, g22, sx[id]);
    complex cr0p11 = crp1 / b2qjvf2[id];
    complex result;
    //01 contribution
    result = wu[id] * crp1;
    fCF_real[id] =result.x; 
    fCF_imag[id] =result.y; 
    result = wu[id+numbers] * crp1;
    fCF_real[id+numbers] =result.x; 
    fCF_imag[id+numbers] =result.y; 

    //02 contribution
    result = w0p22[id] * cr0p11;
    fCF_real[id+numbers*2] =result.x; 
    fCF_imag[id+numbers*2] =result.y; 
    result = w0p22[id+numbers] * cr0p11;
    fCF_real[id+numbers*3] =result.x; 
    fCF_imag[id+numbers*3] =result.y;
  }

}
__global__ void propogator7(
                            double mass,
                            double width,
                            double *sv2,
                            double *sv3,
                            double *b1qjv2,
                            double *b1qbv2,
                            double *b1qjv3,
                            double *b1qbv3,
                            double *w1m12,
                            double *w1m13,
                            double *fCF_real,
                            double *fCF_imag,
                            int numbers)
{
  int id = threadIdx.x+blockDim.x*blockIdx.x;
  if(id<numbers){
    complex crp1 = propogator(mass, width, sv2[id]);
    complex crp11 = propogator(mass, width, sv3[id]);
    complex cr1m12_1 = crp1 / b1qjv2[id] / b1qbv2[id];
    complex cr1m13_1 = crp11 / b1qjv3[id] / b1qbv3[id];
    complex result;
    //1-__1 contribution

    result = w1m12[id] * cr1m12_1 + w1m13[id] * cr1m13_1;
    fCF_real[id] =result.x; 
    fCF_imag[id] =result.y; 
    result = w1m12[id+numbers] * cr1m12_1 + w1m13[id+numbers] * cr1m13_1;
    fCF_real[id+numbers] =result.x; 
    fCF_imag[id+numbers] =result.y; 
  }
}
__global__ void propogator8(
                            double mass,
                            double width,
                            double *sv2,
                            double *sv3,
                            double *b2qbv2,
                            double *b2qbv3,
                            double *b2qjv2,
                            double *b2qjv3,
                            double *w1p12_1,
                            double *w1p13_1,
                            double *w1p12_2,
                            double *w1p13_2,
                            double *w1p12_3,
                            double *w1p13_3,
                            double *w1p12_4,
                            double *w1p13_4,
                            double *fCF_real,
                            double *fCF_imag,
                            int numbers )
{
  int id = threadIdx.x+blockDim.x*blockIdx.x;
  if(id<numbers){
    complex crp1 = propogator(mass, width, sv2[id]);
    complex crp11 = propogator(mass, width, sv3[id]);
    complex c1p12_12 = crp1 / b2qbv2[id];
    complex c1p13_12 = crp11 / b2qbv3[id];
    complex c1p12_13 = crp1 / b2qjv2[id];
    complex c1p13_13 = crp11 / b2qjv3[id];
    complex c1p12_14 = c1p12_12 / b2qjv2[id];
    complex c1p13_14 = c1p13_12 / b2qjv3[id];
    complex result;
    // z 1+ 1
    result = w1p12_1[id] * crp1 + w1p13_1[id] * crp11;
    fCF_real[id] =result.x; 
    fCF_imag[id] =result.y; 
    result = w1p12_1[id+numbers] * crp1 + w1p13_1[id+numbers] * crp11;
    fCF_real[id+numbers] =result.x; 
    fCF_imag[id+numbers] =result.y; 

    // z 1+ 2
    result = w1p12_2[id] * c1p12_12 + w1p13_2[id] * c1p13_12;
    fCF_real[id+numbers*2] =result.x; 
    fCF_imag[id+numbers*2] =result.y; 
    result = w1p12_2[id+numbers] * c1p12_12 + w1p13_2[id+numbers] * c1p13_12;
    fCF_real[id+numbers*3] =result.x; 
    fCF_imag[id+numbers*3] =result.y; 

    // z 1+ 3
    result = w1p12_3[id] * c1p12_13 + w1p13_3[id] * c1p13_13;
    fCF_real[id+numbers*4] =result.x; 
    fCF_imag[id+numbers*4] =result.y; 
    result = w1p12_3[id+numbers] * c1p12_13 + w1p13_3[id+numbers] * c1p13_13;
    fCF_real[id+numbers*5] =result.x; 
    fCF_imag[id+numbers*5] =result.y; 

    // z 1+ 4
    result = w1p12_4[id] * c1p12_14 + w1p13_4[id] * c1p13_14;
    fCF_real[id+numbers*6] =result.x; 
    fCF_imag[id+numbers*6] =result.y; 
    result = w1p12_4[id+numbers] * c1p12_14 + w1p13_4[id+numbers] * c1p13_14;
    fCF_real[id+numbers*7] =result.x; 
    fCF_imag[id+numbers*7] =result.y; 
  }
}
__global__ void propogator6(
                            double mass,
                            double width,
                            double *sx,
                            double *b2qf2xx,
                            double *b2qjvf2,
                            double *b4qjvf2,
                            double *w2p1,
                            double *w2p2,
                            double *w2p3,
                            double *w2p4,
                            double *w2p5,
                            double *fCF_real,
                            double *fCF_imag,
                            int numbers)
{
  int id = threadIdx.x+blockDim.x*blockIdx.x;
  if(id<numbers){
    complex crp1 = propogator1270(mass, width, sx[id]);
    complex cw2p11 = crp1 / b2qf2xx[id];
    complex cw2p12 = cw2p11 / b2qjvf2[id];
    complex cw2p15 = cw2p11 / b4qjvf2[id];
    complex result;
    //21 contribution
    result = w2p1[id] * cw2p11;
    fCF_real[id] =result.x; 
    fCF_imag[id] =result.y; 
    result = w2p1[id+numbers] * cw2p11;
    fCF_real[id+numbers] =result.x; 
    fCF_imag[id+numbers] =result.y; 

    //22 contribution
    result=w2p2[id] * cw2p12;
    fCF_real[id+numbers*2] = result.x;
    fCF_imag[id+numbers*2] = result.y;
    result= w2p2[id+numbers] * cw2p12;
    fCF_real[id+numbers*3]=result.x;
    fCF_imag[id+numbers*3]=result.y;

    //23 contribution
    result = w2p3[id] * cw2p12;
    fCF_real[id+numbers*4] =result.x; 
    fCF_imag[id+numbers*4] =result.y; 
    result = w2p3[id+numbers] * cw2p12;
    fCF_real[id+numbers*5] =result.x; 
    fCF_imag[id+numbers*5] =result.y; 

    //24 contribution
    result = w2p4[id] * cw2p12;
    fCF_real[id+numbers*6] =result.x; 
    fCF_imag[id+numbers*6] =result.y; 
    result = w2p4[id+numbers] * cw2p12;
    fCF_real[id+numbers*7] =result.x; 
    fCF_imag[id+numbers*7] =result.y; 

    //25 contribution
    result = w2p5[id] * cw2p15;
    fCF_real[id+numbers*8] =result.x; 
    fCF_imag[id+numbers*8] =result.y; 
    result = w2p5[id+numbers] * cw2p15;
    fCF_real[id+numbers*9] =result.x; 
    fCF_imag[id+numbers*9] =result.y; 
  }
}
void kernel::calEva()
{
  //cout << "number_of_amplitudes = " << number_of_amplitudes << endl;
  //cout << "number_of_events_ = " << number_of_events_ << endl;
  int i = 0;
  while (i < number_of_amplitudes)
    {
      int propType_now = h_par[end_category * i + propType_category];
      cout << "amplitude i = " << i << endl;
      cout << "propType_now = " << propType_now << endl;
      switch(propType_now)
        {
        case 1: // f0
          {
            double mass0 = h_par[end_category * i + mass_category];
            double width0 = h_par[end_category * i + width_category];
            bool _not_changed =
              ((mass0 == h_par_back[end_category * i + mass_category])
               && (width0 == h_par_back[end_category * i + width_category]));
            if (!_not_changed) {
              cout << "prop = " << propType_now << " : work one time!!!" << endl;
              hipDeviceSynchronize();
              propogator1<<<Blocks,Threads>>>(
                                              mass0,
                                              width0,
                                              d_s23,
                                              d_b2qjvf2,
                                              d_wu,
                                              d_w0p22,
                                              fCF_real+2*i*number_of_data,
                                              fCF_imag+2*i*number_of_data,
                                              number_of_data);
            }
            i = i + 2;
          }
          break;
          //	Flatte   Propagator Contribution
        case 2: // f0 980
          {
            double mass980 = h_par[end_category * i + mass_category];
            double g10 = h_par[end_category * i + g1_category];
            double g20 = h_par[end_category * i + g2_category];
            bool _not_changed =
              ((mass980 == h_par_back[end_category * i + mass_category])
               && (g10 == h_par_back[end_category * i + g1_category])
               && (g20 == h_par_back[end_category * i + g2_category]));
            if (!_not_changed) {
              cout << "prop = " << propType_now << " : work one time!!!" << endl;
              hipDeviceSynchronize();
              propogator2<<<Blocks,Threads>>>(
                                              mass980,
                                              g10,
                                              g20,
                                              d_s23,
                                              d_b2qjvf2,
                                              d_w0p22,
                                              d_wu,
                                              fCF_real+2*i*number_of_data,
                                              fCF_imag+2*i*number_of_data,
                                              number_of_data);
            }
            i = i + 2;
          }
          break;
        case 7: //1m1800
          {
            double mass0 = h_par[end_category * i + mass_category];
            double width0 = h_par[end_category * i + width_category];
            bool _not_changed =
              ((mass0 == h_par_back[end_category * i + mass_category])
               && (width0 == h_par_back[end_category * i + width_category]));
            if (!_not_changed)
              {
                cout << "prop = " << propType_now << " : work one time!!!" << endl;
                hipDeviceSynchronize();
                propogator7<<<Blocks,Threads>>>(
                                                mass0,
                                                width0,
                                                d_sv2,
                                                d_sv3,
                                                d_b1qjv2,
                                                d_b1qbv2,
                                                d_b1qjv3,
                                                d_b1qbv3,
                                                d_w1m12,
                                                d_w1m13,
                                                fCF_real+2*number_of_data*i,
                                                fCF_imag+2*number_of_data*i,
                                                number_of_data);
              }
            i = i + 1;
          }
          break;
        case 8: //1p1800
          {
            double mass0 = h_par[end_category * i + mass_category];
            double width0 = h_par[end_category * i + width_category];
            bool _not_changed =
              ((mass0 == h_par_back[end_category * i + mass_category])
               && (width0 == h_par_back[end_category * i + width_category]));
            if (!_not_changed)
              {
                cout << "prop = " << propType_now << " : work one time!!!" << endl;
                hipDeviceSynchronize();
                propogator8<<<Blocks,Threads>>>(
                                                mass0,
                                                width0,
                                                d_sv2,
                                                d_sv3,
                                                d_b2qbv2,
                                                d_b2qbv3,
                                                d_b2qjv2,
                                                d_b2qjv3,
                                                d_w1p12_1,
                                                d_w1p13_1,
                                                d_w1p12_2,
                                                d_w1p13_2,
                                                d_w1p12_3,
                                                d_w1p13_3,
                                                d_w1p12_4,
                                                d_w1p13_4,
                                                fCF_real+2*i*number_of_data,
                                                fCF_imag+2*i*number_of_data,
                                                number_of_data);
              }
            i = i + 4;
          }
          break;
        case 6: //f2
          {
            double mass0 = h_par[end_category * i + mass_category];
            double width0 = h_par[end_category * i + width_category];
            bool _not_changed =
              ((mass0 == h_par_back[end_category * i + mass_category])
               && (width0 == h_par_back[end_category * i + width_category]));
            if (!_not_changed) {
              cout << "prop = " << propType_now << " : work one time!!!" << endl;
              hipDeviceSynchronize();
              propogator6<<<Blocks,Threads>>>(
                                              mass0,
                                              width0,
                                              d_s23,
                                              d_b2qf2xx,
                                              d_b2qjvf2,
                                              d_b4qjvf2,
                                              d_w2p1,
                                              d_w2p2,
                                              d_w2p3,
                                              d_w2p4,
                                              d_w2p5,
                                              fCF_real+2*i*number_of_data,
                                              fCF_imag+2*i*number_of_data,
                                              number_of_data);
            }
            i = i + 5;
          }
          break;
        default :
          cout << "Do not know how to deal with prop type " << propType_now << endl;
          exit(1);
          ;
        }
    }

  hipDeviceSynchronize();
  cal_fCP<<<1,number_of_amplitudes>>>(d_par,fCP_real,fCP_imag,number_of_amplitudes);
  hipDeviceSynchronize();

}

__global__ void reduce(double *arrays,int numbers,double *result)
{
  int id = threadIdx.x + blockDim.x*blockIdx.x;
  extern __shared__ double s_arrays[];
  if (id< numbers) s_arrays[threadIdx.x]=arrays[id];
  else s_arrays[threadIdx.x]=0;
  __syncthreads();
  for(int i = blockDim.x/2;i>=1;i/=2)
    {
      if(threadIdx.x<i)
      s_arrays[threadIdx.x]+=s_arrays[threadIdx.x+i];
      __syncthreads();
    }
  result[blockIdx.x]=s_arrays[0];
}


__global__ void cal_phsp(double *fCP_real, double *fCP_imag, double *fCF_real, double *fCF_imag, double *result,int number_of_amplitudes,int numbers)
{

  int id=threadIdx.x+blockIdx.x*blockDim.x;
  if(id<numbers)
    {
      complex cw1,cw2,fCP,fCF;
      cw1=cw2=complex(0,0);
      for(int i = 0; i < number_of_amplitudes; i+=1)
        {
          fCP = make_complex(fCP_real[i],fCP_imag[i]);
          fCF = make_complex(fCF_real[id+2*i*numbers],fCF_imag[id+2*i*numbers]);
          cw1 = cw1 + fCP * fCF;

          fCF = make_complex(fCF_real[id+(i*2+1)*numbers],fCF_imag[id+(i*2+1)*numbers]);
  
   
  
          cw2 = cw2 + fCP * fCF;
        }
      
      result[id]=(real(cw1) * real(cw1) + imag(cw1) * imag(cw1) + real(cw2) * real(cw2) + imag(cw2) * imag(cw2)) / 2.0;
    }
}



__global__ void cal_likelihood(double *fCP_real, double *fCP_imag, double *fCF_real, double *fCF_imag, double *fx, int number_of_amplitudes,int numbers)
{
  int id=threadIdx.x+blockIdx.x*blockDim.x;
  if(id<numbers)
    {
      complex cw1,cw2,fCP,fCF;
      cw1=cw2=complex(0,0);
      for(int i = 0; i < number_of_amplitudes; i+=1)
        {
          fCP = make_complex(fCP_real[i],fCP_imag[i]);
          fCF = make_complex(fCF_real[id+2*i*numbers],fCF_imag[id+2*i*numbers]);
          cw1 = cw1 + fCP * fCF;

          fCF = make_complex(fCF_real[id+(i*2+1)*numbers],fCF_imag[id+(i*2+1)*numbers]);
          cw2 = cw2 + fCP * fCF;
        }
      fx[id] = -log((real(cw1) * real(cw1) + imag(cw1) * imag(cw1) + real(cw2) * real(cw2) + imag(cw2) * imag(cw2)) / 2.0);
    }
}

__global__ void cal_penalty(double *fCP_real, double *fCP_imag, double *fCF_real, double *fCF_imag,double *result,  int number_of_amplitudes,int numbers)
{

   int id=threadIdx.x+blockIdx.x*blockDim.x;
   complex fCP,fCF,cw1,cw2;
  double temp;
  cw1=cw2=complex(0,0);
  temp=0;
  if(id<numbers)
    {
      for(int i = 0; i < number_of_amplitudes; i+=1)
        {
          fCP = make_complex(fCP_real[i],fCP_imag[i]);
          fCF = make_complex(fCF_real[id+2*i*numbers],fCF_imag[id+2*i*numbers]);
          cw1 = fCP * conj(fCP);
          cw2 = fCF * conj(fCF) / 2.0;
          fCF = make_complex(fCF_real[id+(i*2+1)*numbers],fCF_imag[id+(i*2+1)*numbers]);
          cw2 = cw2 + fCF * conj(fCF) / 2.0;
          temp+= real(cw1) * real(cw2);
        }
      result[id]=temp;
  
    }
}


kernel::kernel(std::vector<double *> Data, int Device_id, int start, int end,int nAmps,int numbers)
{
  h_par_back=new double[nAmps*end_category];
  h_par=new double[nAmps*end_category];
  h_phsp_container=new double[numbers];
  Threads = threads_per_block;
  Blocks = (end - start + Threads -1)/Threads;
  number_of_data = end-start;
  number_of_amplitudes = nAmps;
  CUDA_CALL(hipSetDevice(Device_id));
  CUDA_CALL(hipMalloc((void **)&d_par,number_of_amplitudes*sizeof(double)*end_category));
  CUDA_CALL(hipMalloc((void **)&d_container,(end-start)*sizeof(double)));

  //prop1
  CUDA_CALL(hipMalloc((void **)&d_w0p22, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_wu, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_b2qjvf2, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_s23, (end-start)*sizeof(double)));
  //prop6
  CUDA_CALL(hipMalloc((void **)&d_b2qf2xx, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_b4qjvf2, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w2p1, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w2p2, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w2p3, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w2p4, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w2p5, 2*(end-start)*sizeof(double)));
  //prop7
  CUDA_CALL(hipMalloc((void **)&d_sv2, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_sv3, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_b1qjv2, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_b1qbv2, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_b1qjv3, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_b1qbv3, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1m12, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1m13, 2*(end-start)*sizeof(double)));
  //prop8
  CUDA_CALL(hipMalloc((void **)&d_b2qbv2, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_b2qbv3, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_b2qjv2, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_b2qjv3, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1p12_1, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1p13_1, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1p12_2, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1p13_2, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1p12_3, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1p13_3, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1p12_4, 2*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_w1p13_4, 2*(end-start)*sizeof(double)));



  CUDA_CALL(hipMalloc((void **)&fCP_real, nAmps*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&fCP_imag, nAmps*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&fCF_real, 2*nAmps*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&fCF_imag, 2*nAmps*(end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_phsp, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_likelihood, (end-start)*sizeof(double)));
  CUDA_CALL(hipMalloc((void **)&d_penalty, (end-start)*sizeof(double)));


  //prop1
  CUDA_CALL(hipMemcpyAsync(d_b2qjvf2, Data[b2qjvf2]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpyAsync(d_wu, Data[wu]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_wu+(end-start), Data[wu]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w0p22, Data[w0p22]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w0p22+(end-start), Data[w0p22]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_s23, Data[s23]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));

  //prop6
  CUDA_CALL(hipMemcpyAsync(d_b2qf2xx, Data[b2qf2xx]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_b4qjvf2, Data[b4qjvf2]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p1, Data[w2p1]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p1+(end-start), Data[w2p1]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p2, Data[w2p2]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p2+(end-start), Data[w2p2]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p3, Data[w2p3]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p3+(end-start), Data[w2p3]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p4, Data[w2p4]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p4+(end-start), Data[w2p4]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p5, Data[w2p5]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w2p5+(end-start), Data[w2p5]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));

  //prop7
  CUDA_CALL(hipMemcpyAsync(d_sv3, Data[sv3]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_sv2, Data[sv2]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_b1qjv2, Data[b1qjv2]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_b1qjv3, Data[b1qjv3]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_b1qbv2, Data[b1qbv2]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_b1qbv3, Data[b1qbv3]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));

  CUDA_CALL(hipMemcpyAsync(d_w1m12, Data[w1m12]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1m12+(end-start), Data[w1m12]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1m13, Data[w1m13]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1m13+(end-start), Data[w1m13]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));


  //prop8
  CUDA_CALL(hipMemcpyAsync(d_b2qbv2, Data[b2qbv2]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_b2qbv3, Data[b2qbv3]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_b2qjv2, Data[b2qjv2]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_b2qjv3, Data[b2qjv3]+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));


  CUDA_CALL(hipMemcpyAsync(d_w1p12_1, Data[w1p12_1]+start, 2*(end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p12_1+(end-start), Data[w1p12_1]+numbers+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p13_1, Data[w1p13_1]+start, 2*(end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p13_1+(end-start), Data[w1p13_1]+numbers+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p12_2, Data[w1p12_2]+start, 2*(end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p12_2+(end-start), Data[w1p12_2]+numbers+start, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p13_2, Data[w1p13_2]+start, 2*(end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p13_2+(end-start), Data[w1p13_2]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p12_3, Data[w1p12_3]+start, 2*(end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p12_3+(end-start), Data[w1p12_3]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p13_3, Data[w1p13_3]+start, 2*(end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p13_3+(end-start), Data[w1p13_3]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p12_4, Data[w1p12_4]+start, 2*(end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p12_4+(end-start), Data[w1p12_4]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p13_4, Data[w1p13_4]+start, 2*(end-start)*sizeof(double),hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyAsync(d_w1p13_4+(end-start), Data[w1p13_4]+start+numbers, (end-start)*sizeof(double),hipMemcpyHostToDevice));
  hipDeviceSynchronize();


 
  

}

void kernel::par_trans(const std::vector<double> par) const
{
  cout<<"number_of_amplitudes: "<<number_of_amplitudes<<endl;
  CUDA_CALL(hipMemcpyAsync(d_par, &par[0],end_category*number_of_amplitudes*sizeof(double),hipMemcpyHostToDevice));
  for(int i=0;i<number_of_amplitudes*end_category;i++)
    {
      h_par[i]=par[i];
    }
}


kernel::kernel()
{
}

double kernel::sum_penalty()
{
  cal_penalty<<<Blocks,Threads>>>(fCP_real, fCP_imag, fCF_real,fCF_imag,d_penalty,  number_of_amplitudes, number_of_data);
  int count=number_of_data;
    while(1)
    {
      reduce<<<(count+Threads-1)/Threads,Threads,Threads*sizeof(double)>>>(d_penalty,count,d_container);
      count=(count+Threads-1)/Threads;
      
      if(count==1)
        {
          hipMemcpyAsync(&h_penalty,d_container,sizeof(double),hipMemcpyDeviceToHost);
          break;
        }
      reduce<<<(count+Threads-1)/Threads,Threads,Threads*sizeof(double)>>>(d_container,count,d_penalty);
      count=(count+Threads-1)/Threads;
      if(count==1)
        {
          hipMemcpyAsync(&h_penalty,d_penalty,sizeof(double),hipMemcpyDeviceToHost);
          break;
        }

        }
    return h_penalty;
}

double kernel::sum_phsp()
{
    cal_phsp<<<Blocks,Threads>>>(fCP_real, fCP_imag, fCF_real,fCF_imag,d_phsp,number_of_amplitudes,number_of_data);
  int count=number_of_data;
     while(1)
    {
      reduce<<<(count+Threads-1)/Threads,Threads,Threads*sizeof(double)>>>(d_phsp,count,d_container);
      count=(count+Threads-1)/Threads;
      
      if(count==1)
        {
          hipMemcpyAsync(&h_phsp,d_container,sizeof(double),hipMemcpyDeviceToHost);
          break;
        }
      reduce<<<(count+Threads-1)/Threads,Threads,Threads*sizeof(double)>>>(d_container,count,d_phsp);
      count=(count+Threads-1)/Threads;
      if(count==1)
        {
          hipMemcpyAsync(&h_phsp,d_phsp,sizeof(double),hipMemcpyDeviceToHost);
          break;
        }
      
        }
     return h_phsp;
}

double  kernel::sum_likelihood()
{
  cal_likelihood<<<Blocks,Threads>>>(fCP_real, fCP_imag, fCF_real,fCF_imag,d_likelihood, number_of_amplitudes,number_of_data);
   int count=number_of_data;
     while(1)
    {
      reduce<<<(count+Threads-1)/Threads,Threads,Threads*sizeof(double)>>>(d_likelihood,count,d_container);
      count=(count+Threads-1)/Threads;
      
      if(count==1)
        {
          hipMemcpyAsync(&h_likelihood,d_container,sizeof(double),hipMemcpyDeviceToHost);
          break;
        }
      reduce<<<(count+Threads-1)/Threads,Threads,Threads*sizeof(double)>>>(d_container,count,d_likelihood);
      count=(count+Threads-1)/Threads;
      if(count==1)
        {
          hipMemcpyAsync(&h_likelihood,d_likelihood,sizeof(double),hipMemcpyDeviceToHost);
          break;
        }

        }
     return h_likelihood;
}
 
void kernel::trans_phsp()
{
    cal_phsp<<<Blocks,Threads>>>(fCP_real, fCP_imag, fCF_real,fCF_imag,d_phsp,number_of_amplitudes,number_of_data);
    hipMemcpyAsync(h_phsp_container,d_phsp,number_of_data*sizeof(double),hipMemcpyDeviceToHost);

}

